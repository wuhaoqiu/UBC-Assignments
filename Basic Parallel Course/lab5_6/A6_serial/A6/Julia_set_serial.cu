#include "hip/hip_runtime.h"

/* Julia_set_serial.cu
*  Created on: Mar 3, 2018
*      Julia set code by Abdallah Mohamed
*      Other files by EasyBMP (see BSD_(revised)_license.txt)
*/


#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "EasyBMP.h"

//Complex number definition
struct Complex {	// typedef is not required for C++
	float x; 		// real part is represented on x-axis in output image
	float y; 		// imaginary part is represented by y-axis in output image
};

//Function declarations
__global__ void compute_julia(uchar4*, int, int);
void save_image(uchar4*, const char*, int, int);
__device__ Complex add(Complex, Complex);
__device__ Complex mul(Complex, Complex);
__device__ float mag(Complex);

//main function
int main(void) {
	char* name = "test.bmp";
	//set the size of block and the grid, make them two dimension
	int width = 3000;
	int height = 3000;
	
	int TILE_WIDTH = 30;
	int TILE_HEIGHT = 30;
	dim3 blocksize(TILE_WIDTH, TILE_HEIGHT);

	int nblk_x = (width - 1) / TILE_WIDTH + 1;
	int nblk_y = (height - 1) / TILE_HEIGHT + 1;
	dim3 gridsize(nblk_x, nblk_y);
	// end of setting

	//create output image
	uchar4 *pixels = (uchar4*)malloc(width * height * sizeof(uchar4));	//uchar4 is a CUDA type representing a vector of 4 chars
	uchar4 *d_pixels;
	hipMalloc(&d_pixels, width * height * sizeof(uchar4));

	compute_julia <<< gridsize,blocksize >>> (d_pixels, width, height);	//width x height
	
	hipMemcpy(pixels, d_pixels, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
																	//Write output image to a file (DO NOT parallelize this function)
	save_image(pixels, name, width, height);

	//free memory
	free(pixels);
	hipFree(d_pixels);
	
	printf("Finished creating %s.\n", name);
	return 0;
}

// serial implementation of Julia set
__global__ void compute_julia(uchar4 *pixels, int width, int height) {
	
	//PROBLEM SETTINGS (marked by '******')
	// **** Accuracy ****: lower values give less accuracy but faster performance
	int max_iterations = 400;
	int infinity = 20;													//used to check if z goes towards infinity

	// ***** Shape ****: other values produce different patterns. See https://en.wikipedia.org/wiki/Julia_set
	Complex c = { 0.285, 0.01 }; 										//the constant in z = z^2 + c

	// ***** Size ****: higher w means smaller size
	float w = 4;
	float h = w * height / width;										//preserve aspect ratio

	// LIMITS for each pixel
	float x_min = -w / 2, y_min = -h / 2;
	float x_incr = w / width, y_incr = h / height;
	
	//****************************************************
	//REQ: Parallelize the following for loop using CUDA 
	//****************************************************



			int col = blockIdx.x*blockDim.x + threadIdx.x;
			int row = blockIdx.y*blockDim.y + threadIdx.y;

			Complex z;
			z.x = x_min + col * x_incr;
			z.y = y_min + row * y_incr;

			
			//iteratively compute z = z^2 + c and check if z goes to infinity
			int n = 0;
			do{
				z = add(mul(z, z), c);								// z = z^2 + c
			} while (mag(z) < infinity && n++ < max_iterations);	// keep looping until z->infinity or we reach max_iterations
			
			// color each pixel based on above loop
			if (n == max_iterations) {								// if we reach max_iterations before z reaches infinity, pixel is black 
				pixels[col + row * width] = { 0,0,0,0 };
			} else {												// if z reaches infinity, pixel color is based on how long it takes z to go to infinity
				unsigned char hue = (unsigned char)(255 * sqrt((float)n / max_iterations));
				pixels[col + row * width] = { hue,hue,hue,255 };
			}
		
	
	

}

void save_image(uchar4* pixels, const char* filename, int width, int height) {
	BMP output;
	output.SetSize(width, height);
	output.SetBitDepth(24);
	// save each pixel to output image
	for (int row = 0; row < height; row++) {
		for (int col = 0; col < width; col++) {
			uchar4 color = pixels[col + row * width];
			output(col, row)->Red = color.x;
			output(col, row)->Green = color.y;
			output(col, row)->Blue = color.z;
		}
	}
	output.WriteToFile(filename);
}

Complex add(Complex c1, Complex c2) {
	return{ c1.x + c2.x, c1.y + c2.y };
}

Complex mul(Complex c1, Complex c2) {
	return{ c1.x * c2.x - c1.y * c2.y, c1.x * c2.y + c2.x * c1.y };
}

float mag(Complex c) {
	return (float)sqrt((double)(c.x * c.x + c.y * c.y));
}