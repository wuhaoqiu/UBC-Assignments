
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 10000000

__global__ void addKernel(float *a, int n)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	a[i] = (float)i / n;
}

int main()
{
	float* a;

	float* d_a;

	a = (float*)malloc(N * sizeof(float));

	hipMalloc(&d_a, N * sizeof(float));

	hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

	int numBlock = N / 1000;

	addKernel <<<numBlock, 1000 >>> (d_a, N);

	printf("a[%d]:%.7f\n", 0, a[0]);
	printf("a[%d]:%.7f\n", 1, a[1]);

	hipMemcpy(a, d_a, N * sizeof(float), hipMemcpyDeviceToHost);

	int n = N;

	printf("a[%d]:%.7f\n", 0, a[0]);
	printf("a[%d]:%.7f\n", 1, a[1]);
	printf("a[%d]:%.7f\n", 2, a[2]);
	printf("a[%d]:%.7f\n", 3, a[3]);
	printf("a[%d]:%.7f\n", 4, a[4]);
	printf("a[%d]:%.7f\n", n - 5, a[n - 5]);
	printf("a[%d]:%.7f\n", n - 4, a[n - 4]);
	printf("a[%d]:%.7f\n", n - 3, a[n - 3]);
	printf("a[%d]:%.7f\n", n - 2, a[n - 2]);
	printf("a[%d]:%.7f\n", n - 1, a[n - 1]);

	free(a);

	hipFree(d_a);

	return 0;
}