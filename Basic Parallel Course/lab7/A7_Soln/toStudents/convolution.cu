#include "hip/hip_runtime.h"
#include "EasyBMP.h"
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>	// for uchar4 struct

#include ""

#define MIN(x,y) (  (y) ^ (((x) ^ (y)) & -((x) < (y))) )
#define MAX(x,y) (  (x) ^ (((x) ^ (y)) & -((x) < (y))) )

#define CHK(call) {	hipError_t err = call;					\
					if (err != hipSuccess) {				\
						printf("Error:code %d, Line %d\n",err,  __LINE__); \
						printf(hipGetErrorString(err));	\
						hipDeviceReset();exit(1);			\
					}										\
				  }


//****************************************************************************************************************
//                                            Serial implementation 
//****************************************************************************************************************

//this function applies the convolution kernel (denoted by filter) to every pixel of the input image (image_in)
//constraints: image_in and image_out have 8-bit pixels (e.g., grayscale images, only one color channel, etc)
void convolution_8bits(const unsigned char* const image_in, unsigned char* const image_out, const int height, const int width, const float *filter, const int filter_width){
	//Apply the filter to every image pixel (col, row) 
	for (int row = 0; row < height; ++row) 
		for (int col = 0; col < width; ++col) {
			float sum = 0.0f;
			//Having the filter centered at image pixel (col,row), multiply every filter pixel by the corresponding image pixel, and find the sum
			for (int row_f = -filter_width / 2; row_f <= filter_width / 2; ++row_f)
				for (int col_f = -filter_width / 2; col_f <= filter_width / 2; ++col_f) {
					//get the value of the image pixel for the current filter pixel. If out of boundary, use boundary pixels
					int row_i = MIN(MAX(row + row_f, 0), (height - 1));
					int col_i = MIN(MAX(col + col_f, 0), (width - 1));
					float pxl_image = image_in[row_i * width + col_i];
					//get the value for the current filter 
					float pxl_filter = filter[(row_f + filter_width / 2) * filter_width + col_f + filter_width / 2];
					//multiply image_pixel by filter_pixel and add to final value of the image pixel
					sum += pxl_image * pxl_filter;
				}
			

			//final value of image pixel(col, row) = the sum of every filter pixel multiplied by the corresponding image pixels around image pixel(col,row)
			image_out[row * width + col] = sum;
		}
}

//This function applies the convolution kernel (denoted by filter) to every pixel of the input image (image_in)
//Constraints:	- Both image_in and image_out are in RGBA formate (32-bit pixels as uchar4)
//				- Filter is a square matrix (float). The sum of all its values is 1 (normalized)
void convolution_32bits( const uchar4* const image_in, uchar4 *const image_out, int height, int width, const float* const filter, const int filter_width){	
	//break the input image (uchar4 matrix) into 4 channels (four char matrices): Red, Green, Blue, and Alpha
	unsigned char *R_in = new unsigned char[width * height];
	unsigned char *G_in = new unsigned char[width * height];
	unsigned char *B_in = new unsigned char[width * height];
	unsigned char *A_in = new unsigned char[width * height];
	for (int i = 0; i < width * height; ++i) {	//break each pixel in input image
		uchar4 pxl = image_in[i];
		R_in[i] = pxl.x;
		G_in[i] = pxl.y;
		B_in[i] = pxl.z;
		A_in[i] = pxl.w;
	}
	//perform 8-bit convolution for each 8-bit image channel 
	unsigned char *R_out = new unsigned char[width * height];
	convolution_8bits(R_in, R_out, height, width, filter, filter_width);

	unsigned char *G_out = new unsigned char[width * height];
	convolution_8bits(G_in, G_out, height, width, filter, filter_width);

	unsigned char *B_out = new unsigned char[width * height];
	convolution_8bits(B_in, B_out, height, width, filter, filter_width);

	unsigned char *A_out = new unsigned char[width * height];
	convolution_8bits(A_in, A_out, height, width, filter, filter_width);

	//merge the four channels into one output image of type uchar4
	for (size_t i = 0; i < height * width; ++i) 
		image_out[i] = make_uchar4(R_out[i], G_out[i], B_out[i], A_out[i]);	
}

//****************************************************************************************************************
//                                            CUDA implementation 
//****************************************************************************************************************

__device__ void convolution_8bits_parallel(int col, int row, const unsigned char* const image_in, unsigned char* const image_out, const int height, const int width, const float *filter, const int filter_width) {
	float sum = 0.0f;
	//Having the filter centered at image pixel (col,row), multiply every filter pixel by the corresponding image pixel, and find the sum
	for (int row_f = -filter_width / 2; row_f <= filter_width / 2; ++row_f)
		for (int col_f = -filter_width / 2; col_f <= filter_width / 2; ++col_f) {
			//get the value of the image pixel for the current filter pixel. If out of boundary, use boundary pixels
			int row_i = MIN(MAX(row + row_f, 0), (height - 1));
			int col_i = MIN(MAX(col + col_f, 0), (width - 1));
			float pxl_image = image_in[row_i * width + col_i];
			//get the value for the current filter 
			float pxl_filter = filter[(row_f + filter_width / 2) * filter_width + col_f + filter_width / 2];
			//multiply image_pixel by filter_pixel and add to final value of the image pixel
			sum += pxl_image * pxl_filter;
		}
	//final value of image pixel(col, row) = the sum of every filter pixel multiplied by the corresponding image pixels around image pixel(col,row)
	image_out[row * width + col] = sum;
}

__global__ void convolution_32bits_parallel_phase1(const uchar4* const image_in, int height, int width, unsigned char *R_in, unsigned char *G_in, unsigned char *B_in, unsigned char *A_in) {
	//Set id's and index variables
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = col + row * width;
	if (col< width && row < height) {
		//break up each pixel in input image
		uchar4 px = image_in[idx];
		R_in[idx] = px.x;
		G_in[idx] = px.y;
		B_in[idx] = px.z;
		A_in[idx] = px.w;
	}
}

__global__ void convolution_32bits_parallel_phase2(uchar4 *const image_out, int height, int width, const float* const filter, const int filter_width,
	unsigned char *R_in, unsigned char *G_in, unsigned char *B_in, unsigned char *A_in,
	unsigned char *R_out, unsigned char *G_out, unsigned char *B_out, unsigned char *A_out) {
	//Set id's and index variables
	int col = blockDim.x * blockIdx.x + threadIdx.x;
	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int idx = col + row * width;
	if (col< width && row < height) {
		//perform 8-bit convolution for each 8-bit image channel 
		convolution_8bits_parallel(col, row, R_in, R_out, height, width, filter, filter_width);
		convolution_8bits_parallel(col, row, G_in, G_out, height, width, filter, filter_width);
		convolution_8bits_parallel(col, row, B_in, B_out, height, width, filter, filter_width);
		convolution_8bits_parallel(col, row, A_in, A_out, height, width, filter, filter_width);

		//merge the four channels into one output image of type uchar4
		image_out[idx] = make_uchar4(R_out[idx], G_out[idx], B_out[idx], A_out[idx]);
	}
}

//**************************************************************
//No need to parallelize any of the functions below this comment
//**************************************************************

//This function reads a BMP image using the EasyBMP library and returns a 1D array representing the RGBA values of the image pixels
//image_out->x is Red, image_out->y is Green, image_out->z is Blue, image_out->w is Alpha
//how to use:	1- in the calling function, declare these variables:	uchar4* img = NULL;	int width = 0, height = 0;
//				2- then call this function								readBMP(filename, &img, &width, &height); 
void readBMP(const char* FileName, uchar4 **image_out, int* width, int* height){
	BMP img;
	img.ReadFromFile(FileName);
	*width = img.TellWidth();
	*height = img.TellHeight();
	uchar4 *const img_uchar4 = (uchar4*)malloc(*width * *height * sizeof(int));
	// save each pixel to image_out as uchar4 in row-major format
	for (int row = 0; row <*height; row++)
		for (int col = 0; col < *width; col++)
			img_uchar4[col + row * *width] = make_uchar4(img(col, row)->Red, img(col, row)->Green, img(col, row)->Blue, img(col, row)->Alpha);	//use row-major
	*image_out = img_uchar4;
}

//This function writes a BMP image using the EasyBMP library
//how to use: in the calling function, call		writeBMP(destination_filename, source_image_array, width, height); 
void writeBMP(const char* FileName, uchar4 *image, int width, int height){
	BMP output;
	output.SetSize(width, height);
	output.SetBitDepth(24);
	// save each pixel to the output image
	for (int row = 0; row < height; row++){		//for each row
		for (int col = 0; col <  width; col++){	//for each col
			uchar4 rgba = image[col + row * width];
			output(col, row)->Red = rgba.x;
			output(col, row)->Green = rgba.y;
			output(col, row)->Blue = rgba.z;
			output(col, row)->Alpha = rgba.w;
		}
	}
	output.WriteToFile(FileName);

}

//Normalize image filter (sum of all values should be 1) 
// the filter is a 2D float array
void normalizeFilter(float* filter, int width){
	//find the sum
	float sum = 0;
	for (int i = 0; i < width*width; i++)
		sum += filter[i];
	//normalize
	for (int i = 0; i < width*width; i++)
		filter[i] /= sum;
}

//this Function reads the convolution-filter image 
//Constrating: Filter is 32 bit RGPA image. The filter must be sqaure. Filter width must be an odd number 
float* readFilter(const char* filter_image_name, int* filter_width){
	int filterHeight;	//for testing that height = width
	//read filter image as 32 bit RGPA bitmap and check the constraints (square, odd width)
	uchar4* filterImageUchar;
	readBMP(filter_image_name, &filterImageUchar, filter_width, &filterHeight);
	if (*filter_width != filterHeight || *filter_width % 2 == 0){
		fprintf(stderr, "Non-square filters or filters with even width are not supported yet. Program terminated!\n");
		exit(1);
	}
	//convert every pixel to a float number representing its grayscale intensity. Formula used is 0.21 R + 0.72 G + 0.07 B
	float* filter = (float*)malloc(*filter_width * *filter_width * sizeof(float));
	for (int i = 0; i < *filter_width * *filter_width; i++){
		uchar4 element = filterImageUchar[i];
		filter[i] = 0.21 * element.x + 0.72 * element.y + 0.07 * element.z; 
	}
	//Normalization makes sure that the sum of all values in the filter is 1 
	normalizeFilter(filter, *filter_width);
	//return result
	return filter;
}


//serial execution
void serial() {
	int filter_width;
	const char* filter_name = "filter_blur_21.bmp";									//filter width = 21 pixels
	const char* image_in_name = "okanagan.bmp";
	const char* image_out_name = "okanagan_blur.bmp";

	//load filter
	float* filter = readFilter(filter_name, &filter_width);
	printf("Filter loaded...\n");

	//load input image
	int width, height;
	uchar4* image_in;
	readBMP(image_in_name, &image_in, &width, &height);								//image_in will have all pixel information, each pixel as uchar4
	printf("Input image loaded...\n");

	//apply convolution filter to input image
	uchar4* image_out = (uchar4*)malloc(width*height * sizeof(uchar4));				//reserve space in the memory for the output image
	printf("Applying the convolution filter...\n");
	int t = clock();
	convolution_32bits(image_in, image_out, height, width, filter, filter_width);	//filter applied to image_in, results saved in image_out
	t = (clock() - t) * 1000 / CLOCKS_PER_SEC;
	printf("Convolution filter applied. Time taken: %d.%d seconds\n", t / 1000, t % 1000);

	//save results to output image
	writeBMP(image_out_name, image_out, width, height);
	printf("Output image saved.\nProgram finished!\n");
}
void parallel() {
	int filter_width;
	const char* filter_image_name = "filter_blur_21.bmp";							//filter width = 21 pixels
	const char* image_in_name = "okanagan.bmp";
	const char* image_out_name = "okanagan_cuda_blur.bmp";

	//load filter
	float* filter = readFilter(filter_image_name, &filter_width);
	printf("Filter loaded...\n");

	//load input image
	int width, height;
	uchar4* image_in;
	readBMP(image_in_name, &image_in, &width, &height);								//image_in will have all pixel information, each pixel as uchar4
	int size = width * height;
	printf("Input image loaded...\n");


	//Allocate memory for the image_out and a pointer for the device
	uchar4* image_out = (uchar4*)malloc(size * sizeof(uchar4));						//reserve space in the memory for the output image
	printf("Applying the convolution filter...\n");


	int t = clock();

	//Allocate memory on the GPU
	uchar4 *d_image_out, *d_image_in;											//input and output images
	CHK(hipMalloc((void**)&d_image_in, size * sizeof(uchar4)));
	CHK(hipMalloc((void**)&d_image_out, size * sizeof(uchar4)));
	
	float* d_filter;																//filter
	CHK(hipMalloc((void**)&d_filter, filter_width * filter_width * sizeof(float)));

	unsigned char *d_R_in,  *d_G_in,  *d_B_in,  *d_A_in,					//4 channels in input image
	unsigned char *d_R_out, *d_G_out, *d_B_out, *d_A_out;					//4 channels in output image
	CHK(hipMalloc((void**)&d_R_in, size * sizeof(uchar4)); hipMalloc((void**)&d_R_out, size * sizeof(uchar4)));
	CHK(hipMalloc((void**)&d_G_in, size * sizeof(uchar4)); hipMalloc((void**)&d_G_out, size * sizeof(uchar4)));
	CHK(hipMalloc((void**)&d_B_in, size * sizeof(uchar4)); hipMalloc((void**)&d_B_out, size * sizeof(uchar4)));
	CHK(hipMalloc((void**)&d_A_in, size * sizeof(uchar4)); hipMalloc((void**)&d_A_out, size * sizeof(uchar4)));

	//Copy image and filter to GPU
	CHK(hipMemcpy(d_filter, filter, filter_width*filter_width*sizeof(float), hipMemcpyHostToDevice));
	CHK(hipMemcpy(d_image_in, image_in, size * sizeof(uchar4), hipMemcpyHostToDevice));

	//launch the kernel
	int nthreads_h = 32;
	int nthreads_v = 32;
	dim3 blocksize(nthreads_h, nthreads_v);												// assuming max num of threads = 1024
	int nblocks_h = (width - 1) / nthreads_h + 1;
	int nblocks_v = (height - 1) / nthreads_v + 1;
	dim3 gridsize(nblocks_h, nblocks_v);
	printf("Using %d x %d blocks, each with %d x %d threads.\n", nblocks_v, nblocks_h, nthreads_v, nthreads_h);
	
	//Kernel is divided into two functions to allow syncing all threads in the gread:
	//Phase 1: split the input image into 4 channels, then ALL thread will sync on exit (that is why we split into two kernels, i.e. to sync all threads in the grid)
	convolution_32bits_parallel_phase1 <<<gridsize, blocksize >>>(	d_image_in, height, width, d_R_in, d_G_in, d_B_in, d_A_in);
	CHK(hipPeekAtLastError());		//check for kernel launch errors 
	CHK(hipDeviceSynchronize());	//check for asynchronous error in kernel

	//Phase 2: process the 4 channels and writ the output image.
	convolution_32bits_parallel_phase2 <<<gridsize, blocksize >>>(	d_image_out, height, width, d_filter, filter_width,
																	d_R_in, d_G_in, d_B_in, d_A_in, d_R_out, d_G_out, d_B_out, d_A_out);
	CHK(hipPeekAtLastError());		//check for kernel launch errors 
	CHK(hipDeviceSynchronize());	//check for asynchronous error in kernel

	//Copy back the result
	CHK(hipMemcpy(image_out, d_image_out, size * sizeof(uchar4), hipMemcpyDeviceToHost));

	//Free all GPU memory
	CHK(hipFree(d_filter));
	CHK(hipFree(d_image_in)); CHK(hipFree(d_image_out));
	CHK(hipFree(d_R_in));     CHK(hipFree(d_R_out));
	CHK(hipFree(d_G_in));     CHK(hipFree(d_G_out));
	CHK(hipFree(d_B_in));     CHK(hipFree(d_B_out));
	CHK(hipFree(d_A_in));     CHK(hipFree(d_A_out));

	t = (clock() - t) * 1000 / CLOCKS_PER_SEC;
	printf("Convolution filter applied. Time taken: %d.%d seconds\n", t / 1000, t % 1000);

	//save results to output image
	writeBMP(image_out_name, image_out, width, height);
	printf("Output image saved.\nProgram finished!\n");
}

//MAIN
int main(){
	//serial();
	parallel();
}